#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define CALL        true
#define PUT         false
#define MAX_STEPS   366
#define MAX_DIM     2
#define MAX_GRID    500
#define MAX_BASIS   5
#define DISCOUNT(dt, r) (expf(-(r) * (dt)))

typedef struct {
    // Simulation control
    int M_PATHS;       // num Monte Carlo paths
    int N_STEPS;       // num time steps
    float T;           // years to maturity
    float dT;          // time step size

    // Option parameters
    float S0;          // initial asset price
    float K;           // strike price
    float r;           // risk-free rate
    int Q_min, Q_max;  // min and max exercise rights

    // Longstaff-Schwartz
    int N_BASIS;       // num basis functions in regression

    // Quantization
    int N_GRID;        // num grid points per time step
    int N_DIM;         // state space dim: 1 for GBM, 2 for OU

    unsigned long RNG_SEED;

    // AR(1) 
    float alpha1;
    float alpha2;
    float sigma1;
    float sigma2;
} SimulationParams;

typedef struct {
    float X0[MAX_DIM];                          // initial state
    float A[MAX_STEPS][MAX_DIM][MAX_DIM];       // A_k transition matrices
    float Tmat[MAX_STEPS][MAX_DIM][MAX_DIM];    // T_k volatility matrices
} AR1ModelParams;

__global__ void init_rng_kernel(hiprandState *states, unsigned long seed, int M) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < M)
        hiprand_init(seed, tid, 0, &states[tid]);
}

__global__ void simulate_paths_kernel(
    float *d_paths,
    const AR1ModelParams *d_ar1,
    hiprandState *d_rng_states,
    const int M_PATHS,
    const int N_STEPS
) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= M_PATHS) return;

    const int N_DIM = 2;
    float X_curr[N_DIM];
    float X_next[N_DIM];

    // load initial state
    for (int d = 0; d < N_DIM; d++) {
        X_curr[d] = d_ar1->X0[d];
        d_paths[(d * N_STEPS + 0) * M_PATHS + tid] = X_curr[d];
    }

    hiprandState local_state = d_rng_states[tid];

    for (int k = 0; k < N_STEPS - 1; k++) {
        float Z[N_DIM];
        for (int d = 0; d < N_DIM; d++) {
            Z[d] = hiprand_normal(&local_state);
        }

        // X_{k+1} = A_k * X_k + T_k * Z
        for (int i = 0; i < N_DIM; i++) {
            float sum = 0.0f;
            for (int j = 0; j < N_DIM; j++) {
                sum += d_ar1->A[k][i][j] * X_curr[j];
            }
            for (int j = 0; j < N_DIM; j++) {
                sum += d_ar1->Tmat[k][i][j] * Z[j];
            }
            X_next[i] = sum;
        }

        for (int d = 0; d < N_DIM; d++) {
            d_paths[(d * N_STEPS + (k + 1)) * M_PATHS + tid] = X_next[d];
            X_curr[d] = X_next[d];
        }
    }

    d_rng_states[tid] = local_state;
}

static float payoff(float S, float K) {
    return fmaxf(K - S, 0.0f);  // put
}

static void evaluate_basis(float x, int N_BASIS, float* basis_out) {
    // Laguerre polynomials
    basis_out[0] = 1.0f;
    if (N_BASIS > 1) basis_out[1] = 1.0f - x;
    for (int i = 2; i < N_BASIS; i++) {
        basis_out[i] = ((2 * i - 1 - x) * basis_out[i - 1] - (i - 1) * basis_out[i - 2]) / i;
    }
}

float price_LSM(float *h_paths, const SimulationParams *sim) {
    int M = sim->M_PATHS;
    int N = sim->N_STEPS;
    int N_BASIS = sim->N_BASIS;
    float K = sim->K;
    float dT = sim->dT;
    float r = sim->r;

    float* cashflows = (float*)calloc(M * N, sizeof(float));
    bool* exercised = (bool*)calloc(M * N, sizeof(bool));
    int* exercise_count = (int*)calloc(M, sizeof(int));

    // initialize with final payoffs at maturity
    int t_final = N - 1;
    for (int m = 0; m < M; m++) {
        float S = h_paths[(0 * N + t_final) * M + m];
        float val = payoff(S, K);
        cashflows[t_final * M + m] = val;
    }

    // backward induction
    for (int t = N - 2; t >= 0; t--) {
        // paths still eligible to exercise
        int n_train = 0;
        float *X = (float*)malloc(sizeof(float) * M);
        float *Y = (float*)malloc(sizeof(float) * M);

        for (int m = 0; m < M; m++) {
            if (exercise_count[m] >= sim->Q_max) continue;

            float S = h_paths[(0 * N + t) * M + m];
            float immediate = payoff(S, K);
            if (immediate > 0.0f) {
                X[n_train] = S;

                // discounted cashflow
                float discounted = 0.0f;
                for (int t2 = t + 1; t2 < N; t2++) {
                    discounted += DISCOUNT((t2 - t) * dT, r) * cashflows[t2 * M + m];
                }

                Y[n_train] = discounted;
                n_train++;
            }
        }

        // regression
        float **Phi = (float**)malloc(n_train * sizeof(float*));
        float *b = (float*)calloc(N_BASIS, sizeof(float));
        float **A = (float**)calloc(N_BASIS, sizeof(float*));
        for (int i = 0; i < N_BASIS; i++) {
            A[i] = (float*)calloc(N_BASIS, sizeof(float));
        }

        for (int i = 0; i < n_train; i++) {
            Phi[i] = (float*)malloc(N_BASIS * sizeof(float));
            evaluate_basis(X[i], N_BASIS, Phi[i]);
        }

        // A = phi.T * phi and b = phi.T * Y
        for (int i = 0; i < N_BASIS; i++) {
            for (int j = 0; j < N_BASIS; j++) {
                for (int k = 0; k < n_train; k++) {
                    A[i][j] += Phi[k][i] * Phi[k][j];
                }
            }
            for (int k = 0; k < n_train; k++) {
                b[i] += Phi[k][i] * Y[k];
            }
        }

        // A * beta = b using Gaussian elimination
        float *beta = (float*)calloc(N_BASIS, sizeof(float));
        for (int i = 0; i < N_BASIS; i++) beta[i] = b[i];

        // exercise decisions
        for (int m = 0; m < M; m++) {
            if (exercise_count[m] >= sim->Q_max) continue;
            float S = h_paths[(0 * N + t) * M + m];
            float phi[MAX_BASIS];
            evaluate_basis(S, N_BASIS, phi);

            float cont_val = 0.0f;
            for (int i = 0; i < N_BASIS; i++) {
                cont_val += beta[i] * phi[i];
            }

            float immediate = payoff(S, K);
            if (immediate > cont_val) {
                cashflows[t * M + m] = immediate;
                exercised[t * M + m] = true;
                exercise_count[m]++;
            }
        }

        // free mgemory
        for (int i = 0; i < n_train; i++) free(Phi[i]);
        free(Phi); free(X); free(Y); free(beta);
        for (int i = 0; i < N_BASIS; i++) free(A[i]);
        free(A); free(b);
    }

    // Compute present value
    float value = 0.0f;
    for (int m = 0; m < M; m++) {
        for (int t = 0; t < N; t++) {
            float c = cashflows[t * M + m];
            value += DISCOUNT(t * dT, r) * c;
        }
    }
    value /= M;

    free(cashflows);
    free(exercised);
    free(exercise_count);

    return value;
}

void generate_quantization_grids(const SimulationParams *sim, const AR1ModelParams *h_ar1, float *h_Gamma) {
    for (int k = 0; k < sim->N_STEPS; k++) {
        float x_std[2];

        for (int d = 0; d < 2; d++) {
            // std deviation assuming mean 0
            float var = 0.0f;
            for (int j = 0; j < 2; j++) {
                var += h_ar1->Tmat[k][d][j] * h_ar1->Tmat[k][d][j];
            }
            x_std[d] = sqrtf(var);
        }

        // square grid uniformly distributed within 3 std
        int grid_size = sim->N_GRID;
        int side = (int)sqrt(grid_size);
        if (side * side != grid_size) {
            fprintf(stderr, "N_GRID must be a perfect square (e.g., 100, 225, 400)\n");
            exit(1);
        }

        for (int i = 0; i < side; i++) {
            float x1 = -3.0f * x_std[0] + (6.0f * x_std[0]) * i / (side - 1);
            for (int j = 0; j < side; j++) {
                float x2 = -3.0f * x_std[1] + (6.0f * x_std[1]) * j / (side - 1);
                int idx = i * side + j;
                h_Gamma[(k * grid_size + idx) * 2 + 0] = x1;
                h_Gamma[(k * grid_size + idx) * 2 + 1] = x2;
            }
        }
    }
}
  
void run_options_pipeline(const SimulationParams *sim, const AR1ModelParams *h_ar1) {
// memory allocation
    const int m_paths = sim->M_PATHS;
    const int n_steps = sim->N_STEPS;
    const int n_dim = sim->N_DIM;
    const int n_grid = sim->N_GRID;

    size_t path_bytes = sizeof(float) * m_paths * n_steps * n_dim;
    size_t V_bytes = sizeof(float) * m_paths;
    size_t gamma_bytes = sizeof(float) * n_steps * n_grid * n_dim;
    size_t pkij_bytes = sizeof(int) * n_steps * n_grid * n_grid;
    size_t pki_bytes = sizeof(int) * n_steps * n_grid;
    size_t Pkij_bytes = sizeof(float) * n_steps * n_grid * n_grid;

    // host
    float *h_Gamma = NULL; 
    float *h_paths = NULL;

    checkCudaErrors(hipHostMalloc(&h_Gamma, gamma_bytes));
    checkCudaErrors(hipHostMalloc(&h_paths, path_bytes));

    // device
    AR1ModelParams *d_ar1 = NULL;
    float *d_paths = NULL;
    float *d_V = NULL;
    float *d_Gamma = NULL;
    float *d_Pkij = NULL;
    int *d_pkij = NULL;
    int *d_pki = NULL;
    hiprandState *d_rng_states = NULL;

    checkCudaErrors(hipMalloc(&d_ar1, sizeof(AR1ModelParams)));                // ar1 sim for device
    hipMemcpy(d_ar1, h_ar1, sizeof(AR1ModelParams), hipMemcpyHostToDevice);

    checkCudaErrors(hipMalloc(&d_paths, path_bytes));      // simulation paths
    checkCudaErrors(hipMalloc(&d_V, V_bytes));             // value vector
    checkCudaErrors(hipMalloc(&d_Gamma, gamma_bytes));     // quantization grids
    checkCudaErrors(hipMalloc(&d_Pkij, Pkij_bytes));       // transition probabilities

    // transition counters for both quatizations
    checkCudaErrors(hipMalloc(&d_pkij, pkij_bytes));
    checkCudaErrors(hipMalloc(&d_pki, pki_bytes));
    checkCudaErrors(hipMemset(d_pkij, 0, pkij_bytes));
    checkCudaErrors(hipMemset(d_pki, 0, pki_bytes));

    checkCudaErrors(hipMalloc(&d_rng_states, sizeof(hiprandState) * m_paths));      // RNG states

// init quatization grid
    generate_quantization_grids(sim, h_ar1, h_Gamma);
    hipMemcpy(d_Gamma, h_Gamma, gamma_bytes, hipMemcpyHostToDevice);

    dim3 block(256);
    dim3 grid((m_paths + block.x - 1) / block.x);

// init rng
    init_rng_kernel<<<grid, block>>>(d_rng_states, sim->RNG_SEED, m_paths);
    checkCudaErrors(hipDeviceSynchronize());

// Monte Carlo simulation
    simulate_paths_kernel<<<grid, block>>>(
        d_paths, d_ar1, d_rng_states, m_paths, n_steps
    );
    checkCudaErrors(hipDeviceSynchronize());
    hipMemcpy(h_paths, d_paths, path_bytes, hipMemcpyDeviceToHost);

// Longstaff-Schwartz CPU only
    float V_LSM = price_LSM(h_paths, sim);

// to do
    // Tree Quantization Algorithm II
    // estimate_transition_probabilities_algo2();
    // V_TQ2 = backward_induction_quantized();

    // // Run Tree Quantization Algorithm III
    // estimate_transition_probabilities_algo3();
    // V_TQ3 = backward_induction_quantized();

    // print_results(V_LSM, V_TQ2, V_TQ3);
    // compare_performance();

// clean up
    if (h_Gamma) hipHostFree(h_Gamma);

    h_Gamma = NULL;

    if (d_paths) hipFree(d_paths);
    if (d_V) hipFree(d_V);
    if (d_Gamma) hipFree(d_Gamma);
    if (d_pkij) hipFree(d_pkij);
    if (d_pki) hipFree(d_pki);
    if (d_Pkij) hipFree(d_Pkij);
    if (d_rng_states) hipFree(d_rng_states);

    d_paths = NULL;
    d_V = NULL;
    d_Gamma = NULL;
    d_pkij = NULL;
    d_pki = NULL;
    d_Pkij = NULL;
    d_rng_states = NULL;    
}


void printUsage(char *prog) {
    printf("Usage: %s [OPTIONS]\n", prog);
    printf("Options:                                Defaults:\n");
    printf("  -m  Number of Monte Carlo samples  -  100000\n");
    printf("  -n  Number of time steps - - - - - -  365\n");
    printf("  -g  Grid size per time step  - - - -  400\n");
    printf("  -t  Time (years) to maturity - - - -  1.0\n");
    printf("  -s  Initial stock price  - - - - - -  100.0\n");
    printf("  -k  Strike price - - - - - - - - - -  100.0\n");
    printf("  -r  Risk-free rate - - - - - - - - -  0.05\n");
    printf("  -q  Minimum quantity - - - - - - - -  50\n");
    printf("  -Q  Maximum quantity - - - - - - - -  150\n");
    printf("  -S  Random number generator seed - -  42\n");
    printf("  -w First Gaussian weight alpha1 - -  1.0\n");
    printf("  -x Second Gaussian weight alpha2  -  0.4\n");
    printf("  -y First Gaussian sigma1  - - - - -  0.3\n");
    printf("  -z Second Gaussian sigma2 - - - - -  0.2\n");
}

void printSimulationParams(SimulationParams *sim) {
    printf("Model parameters:\n");
    printf("  Number of Monte Carlo samples  -  %d\n", sim->M_PATHS);
    printf("  Number of time steps - - - - - -  %d\n", sim->N_STEPS);
    printf("  Grid size per time step  - - - -  %d\n", sim->N_GRID);
    printf("  Time (years) to maturity - - - -  %.4f\n", sim->T);
    printf("  Initial stock price  - - - - - -  %.4f\n", sim->S0);
    printf("  Strike price - - - - - - - - - -  %.4f\n", sim->K);
    printf("  Risk-free rate - - - - - - - - -  %.4f\n", sim->r);
    printf("  Minimum quantity - - - - - - - -  %d\n", sim->Q_min);
    printf("  Maximum quantity - - - - - - - -  %d\n", sim->Q_max);
    printf("  Random number generator seed - -  %lu\n", sim->RNG_SEED);
    printf("  First Gaussian weight alpha1 - -  %.4f\n", sim->alpha1);
    printf("  Second Gaussian weight alpha2  -  %.4f\n", sim->alpha2);
    printf("  First Gaussian sigma1  - - - - -  %.4f\n", sim->sigma1);
    printf("  Second Gaussian sigma2 - - - - -  %.4f\n", sim->sigma2);
}

void init_SimulationParams(SimulationParams *sim) {
    sim->M_PATHS = 100000;
    sim->N_STEPS = 365;
    sim->T = 1.0f;
    sim->dT = 1.0f / 365;
    sim->S0 = 100.0f;
    sim->K = 100.0f;
    sim->r = 0.05f;
    sim->Q_min = 50;
    sim->Q_max = 150;
    sim->N_BASIS = 3;
    sim->N_GRID = 400;
    sim->N_DIM = 2;
    sim->RNG_SEED = 42UL;
    sim->alpha1 = 1.0f;
    sim->alpha2 = 0.4f;
    sim->sigma1 = 0.3f;
    sim->sigma2 = 0.2f;
}

void init_AR1ModelParams(AR1ModelParams *ar1, SimulationParams *sim) {
    for (int d = 0; d < MAX_DIM; d++) {
        ar1->X0[d] = 0.0f;
    }

    const float dT = sim->dT;
    const float alpha1 = sim->alpha1, alpha2 = sim->alpha2;
    const float sigma1 = sim->sigma1, sigma2 = sim->sigma2;
    const int n_steps = sim->N_STEPS;

    for (int k = 0; k < n_steps; k++) {
        float e1 = expf(-alpha1 * dT);
        float e2 = expf(-alpha2 * dT);

        ar1->A[k][0][0] = e1;
        ar1->A[k][0][1] = 0.0f;
        ar1->A[k][1][0] = 0.0f;
        ar1->A[k][1][1] = e2;

        float var1 = sigma1 * sqrtf((1 - expf(-2.0f * alpha1 * dT)) / (2.0f * alpha1));
        float var2 = sigma2 * sqrtf((1 - expf(-2.0f * alpha2 * dT)) / (2.0f * alpha2));

        ar1->Tmat[k][0][0] = var1;
        ar1->Tmat[k][0][1] = 0.0f;
        ar1->Tmat[k][1][0] = 0.0f;
        ar1->Tmat[k][1][1] = var2;
    }
}


int main(int argc, char **argv) {
    SimulationParams *sim = (SimulationParams*) malloc(sizeof(SimulationParams));
    init_SimulationParams(sim);

    int opt;
    int val_i;
    float val_f;
    unsigned long val_ul;
    while ((opt = getopt(argc, argv, "m:n:g:t:s:k:r:q:Q:S:w:x:y:z:")) != -1) {
        switch (opt) {
            case 'm':
                val_i = atoi(optarg);
                if (val_i > 0) sim->M_PATHS = val_i; 
                break;
            case 'n':             
                val_i = atoi(optarg);
                if (val_i > 0 && val_i < MAX_STEPS) sim->N_STEPS = val_i; 
                break;
            case 'g': 
                val_i = atoi(optarg);
                if (val_i > 0 && val_i < MAX_GRID) sim->N_GRID = val_i; 
                break;
            case 't':
                val_f = atof(optarg);
                if (val_f > 0.0f) sim->T = val_f; 
                break;
            case 's':
                val_f = atof(optarg);
                if (val_f > 0.0f) sim->S0 = val_f; 
                break;
            case 'k':
                val_f = atof(optarg);
                if (val_f > 0.0f) sim->K = val_f; 
                break;
            case 'r':
                val_f = atof(optarg);
                if (val_f > 0.0f) sim->r = val_f; 
                break;
            case 'q':
                val_i = atoi(optarg);
                if (val_i > 0) sim->Q_min = val_i; 
                break;
            case 'Q':
                val_i = atoi(optarg);
                if (val_i > 0) sim->Q_max = val_i; 
                break;
            case 'S':
                val_ul = strtoul(optarg, NULL, 10);
                if (val_ul > 0UL) sim->RNG_SEED = val_ul;
                break;
            case 'w':
                val_f = atof(optarg);
                if (val_f > 0.0f) sim->alpha1 = val_f; 
                break;
            case 'x':
                val_f = atof(optarg);
                if (val_f > 0.0f) sim->alpha2 = val_f; 
                break;
            case 'y':
                val_f = atof(optarg);
                if (val_f > 0.0f) sim->sigma1 = val_f; 
                break;
            case 'z':
                val_f = atof(optarg);
                if (val_f > 0.0f) sim->sigma2 = val_f; 
                break;
            default:
                printUsage(argv[0]);
                free(sim);
                exit(EXIT_FAILURE);
        }
    }

    AR1ModelParams *ar1 = (AR1ModelParams*) malloc(sizeof(AR1ModelParams));
    init_AR1ModelParams(ar1, sim);

    printSimulationParams(sim);

    run_options_pipeline(sim, ar1);

    free(sim);
    free(ar1);
    return EXIT_SUCCESS;
}
